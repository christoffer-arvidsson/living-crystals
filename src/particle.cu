#include "hip/hip_runtime.h"
#include <stdint.h>
#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "particle.h"

#define RNG_SEED 1234

#define LOOP_AROUND true

#define PARTICLES_CAPACITY 64
Particle particles[PARTICLES_CAPACITY];
size_t particles_count = 0;

__global__ void setup_rng(hiprandState* state) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(RNG_SEED, idx, 0, &state[idx]);
}

__global__ void update_state(Particle* particles, size_t n_particles, hiprandState* curand_state) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n_particles) {
        Particle particle = particles[idx];

        // precompute these
        const float trans_coeff = 0.22;
        const float rot_coeff = 0.16;
        const float d_t = 0.1;
        const float sq_trans = sqrtf(2 * trans_coeff);
        const float sq_rot = sqrtf(2 * rot_coeff);

        // normally distributed random numbers
        float weight_x = hiprand_normal(curand_state);
        float weight_y = hiprand_normal(curand_state);
        float weight_rot = hiprand_normal(curand_state);

        // dx(t)/dt = v * cos(theta(t)) + sqrt(2 * D_T) * W_x
        float diff_x = particle.speed * cosf(particle.orient) + sq_trans * weight_x;
        // dy(t)/dt = v * sin(theta(t)) + sqrt(2 * D_T) * W_y
        float diff_y = particle.speed * sinf(particle.orient) + sq_trans * weight_y;
        // dtheta(t)/dt = sqrt(2 * D_R) * W_theta
        float diff_orient = sq_rot * weight_rot;

        particles[idx].p_x += diff_x * d_t;
        particles[idx].p_y += diff_y * d_t;
        particles[idx].orient += diff_orient * d_t;

        #ifdef LOOP_AROUND
        particles[idx].p_x = fmod(particles[idx].p_x, 800.0f);
        particles[idx].p_y = fmod(particles[idx].p_y, 600.0f);
        #endif
    }

}

void clear_particles(void) {
    particles_count = 0;
}

void push_particle(float p_x, float p_y, float speed, float orient) {
    assert(particles_count < PARTICLES_CAPACITY);
    particles[particles_count].p_x = p_x;
    particles[particles_count].p_y = p_y;
    particles[particles_count].speed = speed;
    particles[particles_count].orient = 0.0;
    particles[particles_count].charge = ACTIVE;
    particles[particles_count].radius = 5.0;
    particles_count += 1;
}


void print_particle(Particle* particle) {
    printf("x: %f y: %f speed: %f rad: %f orient: %f\n",
           particle->p_x,
           particle->p_y,
           particle->speed,
           particle->radius,
           particle->orient);
}

Particle* d_particles;
hiprandState *d_state;

void init_simulation(void) {
    // Setup particles
    hipMalloc((void**)&d_particles, particles_count * sizeof(Particle));
    hipMemcpy(d_particles, particles, particles_count * sizeof(Particle), hipMemcpyHostToDevice);

    // Setup rng
    hipMalloc(&d_state, sizeof(hiprandState));
    setup_rng<<<1,1>>>(d_state);
}
void tick_simulation(void) {
    update_state<<<1, particles_count>>>(d_particles, particles_count, d_state);
    hipMemcpy(particles, d_particles, particles_count * sizeof(Particle), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
}

Particle* get_particle(size_t idx) {
    assert (idx < particles_count);
    return &particles[idx];
}

size_t get_num_particles(void) {
    return particles_count;
}

int main_2() {
    // Particles
    clear_particles();
    push_particle(50.0, 50.0, 0.0, 0.0);
    push_particle(-50.0, 50.0, 1.0, 0.0);
    push_particle(-50.0, -50.0, 2.0, 0.0);
    push_particle(50.0, -50.0, 3.0, 0.0);

    init_simulation();

    for (size_t step=0; step < 100; ++step) {
        tick_simulation();
    }

    return 0;
}
